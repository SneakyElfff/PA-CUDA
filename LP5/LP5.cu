#include "hip/hip_runtime.h"
#include <iostream>
#include <opencv2/opencv.hpp>
#include <hip/hip_runtime.h>

using namespace std;
using namespace cv;

__global__ void laplacianFilter(unsigned char *input, unsigned char *output, int width, int height) {
    int x = blockIdx.x * blockDim.x + threadIdx.x;
    int y = blockIdx.y * blockDim.y + threadIdx.y;

    // Проверка, чтобы поток обрабатывал только допустимые координаты
    if (x >= width || y >= height) return;

    if (x > 0 && x < width - 1 && y > 0 && y < height - 1) {
        int idx = y * width + x;

        int laplacian_value = input[(y - 1) * width + x] +    // верхний сосед
                              input[y * width + (x - 1)] +    // левый сосед
                              input[y * width + (x + 1)] +    // правый сосед
                              input[(y + 1) * width + x] +    // нижний сосед
                              -4 * input[idx];               // центральный пиксель

        // Ограничение значений результата в диапазоне [0, 255]
        output[idx] = min(max(laplacian_value, 0), 255);
    } else {
        // Для границ копируется исходное значение
        int idx = y * width + x;
        output[idx] = input[idx];
    }
}

void laplacianFilterCPU(const Mat &input, Mat &output) {
    for (int y = 1; y < input.rows - 1; ++y) {
        for (int x = 1; x < input.cols - 1; ++x) {
            int laplacian_value = input.at<uchar>(y - 1, x) +
                                  input.at<uchar>(y, x - 1) +
                                  input.at<uchar>(y, x + 1) +
                                  input.at<uchar>(y + 1, x) +
                                  -4 * input.at<uchar>(y, x);
            output.at<uchar>(y, x) = min(max(laplacian_value, 0), 255);
        }
    }

    // Копируются граничные пиксели
    for (int y = 0; y < input.rows; ++y) {
        output.at<uchar>(y, 0) = input.at<uchar>(y, 0);          // Левый край
        output.at<uchar>(y, input.cols - 1) = input.at<uchar>(y, input.cols - 1); // Правый край
    }
    for (int x = 0; x < input.cols; ++x) {
        output.at<uchar>(0, x) = input.at<uchar>(0, x);          // Верхний край
        output.at<uchar>(input.rows - 1, x) = input.at<uchar>(input.rows - 1, x); // Нижний край
    }
}

int main(int argc, char** argv) {
    if (argc < 2) {
        cout << "Usage: " << argv[0] << " <image_path>" << endl;
        return -1;
    }

    Mat inputImage = imread(argv[1], IMREAD_GRAYSCALE);
    if (inputImage.empty()) {
        cout << "Could not open or find the image!" << endl;
        return -1;
    }

    int width = inputImage.cols;
    int height = inputImage.rows;

    Mat outputImageCPU = inputImage.clone();
    Mat outputImageGPU = inputImage.clone();

    auto startCPU = chrono::high_resolution_clock::now();
    laplacianFilterCPU(inputImage, outputImageCPU);
    auto endCPU = chrono::high_resolution_clock::now();
    auto durationCPU = chrono::duration_cast<chrono::milliseconds>(endCPU - startCPU).count();
    cout << "CPU processing time: " << durationCPU << " ms" << endl;

    unsigned char *d_input, *d_output;

    hipMalloc(&d_input, width * height * sizeof(unsigned char));
    hipMalloc(&d_output, width * height * sizeof(unsigned char));

    hipMemcpy(d_input, inputImage.data, width * height * sizeof(unsigned char), hipMemcpyHostToDevice);

    dim3 blockSize(16, 16);
    dim3 gridSize((width + blockSize.x - 1) / blockSize.x, (height + blockSize.y - 1) / blockSize.y);
    auto startGPU = chrono::high_resolution_clock::now();
    laplacianFilter<<<gridSize, blockSize>>>(d_input, d_output, width, height);
    hipDeviceSynchronize();
    auto endGPU = chrono::high_resolution_clock::now();
    auto durationGPU = chrono::duration_cast<chrono::milliseconds>(endGPU - startGPU).count();
    cout << "GPU processing time: " << durationGPU << " ms" << endl;

    hipMemcpy(outputImageGPU.data, d_output, width * height * sizeof(unsigned char), hipMemcpyDeviceToHost);

    hipFree(d_input);
    hipFree(d_output);

    imwrite("output_cpu.png", outputImageCPU);
    imwrite("output_gpu.png", outputImageGPU);

    return 0;
}
